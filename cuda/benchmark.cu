#include "hip/hip_runtime.h"
// kernels from http://ppc.cs.aalto.fi/ch4 (2018)
#include <algorithm>
#include <chrono>
#include <iomanip>
#include <iostream>
#include <iostream>
#include <limits>
#include <numeric>
#include <random>

#include <cstdio>
#include <hip/hip_runtime.h>

inline void check(hipError_t err, const char* context) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << context << ": "
            << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

#define CHECK(x) check(x, #x)

#define BLOCKSIZE 32


float next_float() {
    static std::random_device rd;
    static std::default_random_engine e(rd());
    static std::uniform_real_distribution<float> floats(0.0, 1.0);
    return floats(e);
}


inline int static divup(int a, int b) {
    return (a + b - 1)/b;
}


inline int static roundup(int a, int b) {
    return divup(a, b) * b;
}


__global__ void kernel_v0(const float *in, float *out, int n) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    float v = HUGE_VALF;
    for (int k = 0; k < n; ++k) {
        float x = in[n*i + k];
        float y = in[n*k + j];
        float z = x + y;
        v = min(v, z);
    }
    out[n*i + j] = v;
}


__global__ void kernel_v1(const float *in, float *out, int n) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    float v = HUGE_VALF;
    for (int k = 0; k < n; ++k) {
        float x = in[n*j + k];
        float y = in[n*k + i];
        float z = x + y;
        v = min(v, z);
    }
    out[n*j + i] = v;
}


__global__ void kernel_v2(float* r, const float* d, int n, int nn) {
    int ia = threadIdx.x;
    int ja = threadIdx.y;
    int ic = blockIdx.x;
    int jc = blockIdx.y;

    const float* t = d + nn * nn;

    float v[8][8];
    for (int ib = 0; ib < 8; ++ib) {
        for (int jb = 0; jb < 8; ++jb) {
            v[ib][jb] = HUGE_VALF;
        }
    }
    for (int k = 0; k < n; ++k) {
        float x[8];
        float y[8];
        for (int ib = 0; ib < 8; ++ib) {
            int i = ic * 64 + ib * 8 + ia;
            x[ib] = t[nn*k + i];
        }
        for (int jb = 0; jb < 8; ++jb) {
            int j = jc * 64 + jb * 8 + ja;
            y[jb] = d[nn*k + j];
        }
        for (int ib = 0; ib < 8; ++ib) {
            for (int jb = 0; jb < 8; ++jb) {
                v[ib][jb] = min(v[ib][jb], x[ib] + y[jb]);
            }
        }
    }
    for (int ib = 0; ib < 8; ++ib) {
        for (int jb = 0; jb < 8; ++jb) {
            int i = ic * 64 + ib * 8 + ia;
            int j = jc * 64 + jb * 8 + ja;
            if (i < n && j < n) {
                r[n*i + j] = v[ib][jb];
            }
        }
    }
}


__global__ void add_padding_v2(const float* r, float* d, int n, int nn) {
    int ja = threadIdx.x;
    int i = blockIdx.y;

    float* t = d + nn * nn;

    for (int jb = 0; jb < nn; jb += 64) {
        int j = jb + ja;
        float v = (i < n && j < n) ? r[n*i + j] : HUGE_VALF;
        d[nn*i + j] = v;
        t[nn*j + i] = v;
    }
}


__global__ void kernel_fully_coalesced(const float *in, float *out, int n) {
    const float c = 2.0;
    const int i = threadIdx.x + blockIdx.y * blockDim.x;
    float x = input[i];
    out[i] = c * x;
}


__global__ void kernel_poorly_coalesced(const float *in, float *out, int n) {
    const float c = 2.0;
    const int i = n * threadIdx.y + blockIdx.x;
    float x = input[i];
    out[i] = c * x;
}


void step_v0(float* r, const float* d, int n) {
    // Allocate memory & copy data to GPU
    float* dGPU = NULL;
    float* rGPU = NULL;

    CHECK(hipMalloc(&dGPU, n * n * sizeof(float)));
    CHECK(hipMalloc(&rGPU, n * n * sizeof(float)));
    CHECK(hipMemcpy(dGPU, d, n * n * sizeof(float), hipMemcpyHostToDevice));

    // Run kernel
    dim3 dimBlock(16, 16);
    dim3 dimGrid(divup(n, dimBlock.x), divup(n, dimBlock.y));
    kernel_v0<<<dimGrid, dimBlock>>>(rGPU, dGPU, n);
    CHECK(hipGetLastError());

    // Copy data back to CPU & release memory
    CHECK(hipMemcpy(r, rGPU, n * n * sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipFree(dGPU));
    CHECK(hipFree(rGPU));
}


void step_v1(float* r, const float* d, int n) {
    float* dGPU = NULL;
    float* rGPU = NULL;

    CHECK(hipMalloc(&dGPU, n * n * sizeof(float)));
    CHECK(hipMalloc(&rGPU, n * n * sizeof(float)));
    CHECK(hipMemcpy(dGPU, d, n * n * sizeof(float), hipMemcpyHostToDevice));

    dim3 dimBlock(16, 16);
    dim3 dimGrid(divup(n, dimBlock.x), divup(n, dimBlock.y));
    kernel_v1<<<dimGrid, dimBlock>>>(rGPU, dGPU, n);
    CHECK(hipGetLastError());

    CHECK(hipMemcpy(r, rGPU, n * n * sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipFree(dGPU));
    CHECK(hipFree(rGPU));
}


void step_v2(float* r, const float* d, int n) {
    int nn = roundup(n, 64);

    float* dGPU = NULL;
    float* rGPU = NULL;

    CHECK(hipMalloc(&dGPU, 2 * nn * nn * sizeof(float)));
    CHECK(hipMalloc(&rGPU, n * n * sizeof(float)));
    CHECK(hipMemcpy(dGPU, d, n * n * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(rGPU, d, n * n * sizeof(float), hipMemcpyHostToDevice));

    {
        dim3 dimBlock(64, 1);
        dim3 dimGrid(1, nn);
        add_padding_v2<<<dimGrid, dimBlock>>>(rGPU, dGPU, n, nn);
        CHECK(hipGetLastError());
    }

    {
        dim3 dimBlock(8, 8);
        dim3 dimGrid(nn / 64, nn / 64);
        kernel_v2<<<dimGrid, dimBlock>>>(rGPU, dGPU, n, nn);
        CHECK(hipGetLastError());
    }

    CHECK(hipMemcpy(r, rGPU, n * n * sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipFree(dGPU));
    CHECK(hipFree(rGPU));
}


void fully_coalesced(float* r, const float* d, int n) {
    float* dGPU = NULL;
    float* rGPU = NULL;

    CHECK(hipMalloc(&dGPU, n * n * sizeof(float)));
    CHECK(hipMalloc(&rGPU, n * n * sizeof(float)));
    CHECK(hipMemcpy(dGPU, d, n * n * sizeof(float), hipMemcpyHostToDevice));

    dim3 dimBlock(n, 1);
    dim3 dimGrid(1, n);
    kernel_v1<<<dimGrid, dimBlock>>>(rGPU, dGPU, n);
    CHECK(hipGetLastError());

    CHECK(hipMemcpy(r, rGPU, n * n * sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipFree(dGPU));
    CHECK(hipFree(rGPU));
}


void poorly_coalesced(float* r, const float* d, int n) {
    float* dGPU = NULL;
    float* rGPU = NULL;

    CHECK(hipMalloc(&dGPU, n * n * sizeof(float)));
    CHECK(hipMalloc(&rGPU, n * n * sizeof(float)));
    CHECK(hipMemcpy(dGPU, d, n * n * sizeof(float), hipMemcpyHostToDevice));

    dim3 dimBlock(1, n);
    dim3 dimGrid(n, 1);
    kernel_v1<<<dimGrid, dimBlock>>>(rGPU, dGPU, n);
    CHECK(hipGetLastError());

    CHECK(hipMemcpy(r, rGPU, n * n * sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipFree(dGPU));
    CHECK(hipFree(rGPU));
}


void print_header() {
    std::cout << std::setw(8) << "func"
              << std::setw(8) << "iter"
              << std::setw(8) << "n"
              << std::setw(10) << "time (s)"
              << std::endl;
}
void print_row(const char* name, int i, size_t n, double time) {
    std::cout << std::setw(8) << name
              << std::setw(8) << i
              << std::setw(8) << n
              << std::setw(10) << std::setprecision(3) << time
              << std::endl;
}


struct StepFunction {
    const char* name;
    void (*callable)(float*, const float*, int);
};


int main(int argc, char** argv) {
    int iterations = 1;
    if (argc > 1) {
        iterations = std::stoi(argv[1]);
    }

    const size_t n = BLOCKSIZE << 7;
    std::vector<StepFunction> step_functions = {
        {"step_v0", step_v0},
        {"step_v1", step_v1},
        {"step_v2", step_v2},
        {"fully_coalesced", fully_coalesced},
        {"poorly_coalesced", poorly_coalesced},
    };

    print_header();

    for (auto func : step_functions) {
        for (auto i = 0; i < iterations; ++i) {
            std::vector<float> data(n*n);
            std::generate(data.begin(), data.end(), next_float);
            std::vector<float> result(n*n);
            const auto time_start = std::chrono::high_resolution_clock::now();
            func.callable(result.data(), data.data(), n);
            const auto time_end = std::chrono::high_resolution_clock::now();
            const std::chrono::duration<float> delta_seconds = time_end - time_start;
            print_row(func.name, i + 1, n, delta_seconds.count());
        }
    }

}
