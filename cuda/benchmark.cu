// kernels from http://ppc.cs.aalto.fi/ch4 (2018)
#include <algorithm>
#include <chrono>
#include <iomanip>
#include <iostream>
#include <iostream>
#include <limits>
#include <numeric>
#include <random>

#include <cstdio>
#include <hip/hip_runtime.h>

inline void check(hipError_t err, const char* context) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << context << ": "
            << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

#define CHECK(x) check(x, #x)

#define BLOCKSIZE 32


float next_float() {
    static std::random_device rd;
    static std::default_random_engine e(rd());
    static std::uniform_real_distribution<float> floats(0.0, 1.0);
    return floats(e);
}


inline int static divup(int a, int b) {
    return (a + b - 1)/b;
}


inline int static roundup(int a, int b) {
    return divup(a, b) * b;
}


__global__ void kernel_v0(const float *in, float *out, int n) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    float v = HUGE_VALF;
    for (int k = 0; k < n; ++k) {
        float x = in[n*i + k];
        float y = in[n*k + j];
        float z = x + y;
        v = min(v, z);
    }
    out[n*i + j] = v;
}


__global__ void kernel_v1(const float *in, float *out, int n) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    float v = HUGE_VALF;
    for (int k = 0; k < n; ++k) {
        float x = in[n*j + k];
        float y = in[n*k + i];
        float z = x + y;
        v = min(v, z);
    }
    out[n*j + i] = v;
}


__global__ void kernel_v2(float* r, const float* d, int n, int nn) {
    int ia = threadIdx.x;
    int ja = threadIdx.y;
    int ic = blockIdx.x;
    int jc = blockIdx.y;

    const float* t = d + nn * nn;

    float v[8][8];
    for (int ib = 0; ib < 8; ++ib) {
        for (int jb = 0; jb < 8; ++jb) {
            v[ib][jb] = HUGE_VALF;
        }
    }
    for (int k = 0; k < n; ++k) {
        float x[8];
        float y[8];
        for (int ib = 0; ib < 8; ++ib) {
            int i = ic * 64 + ib * 8 + ia;
            x[ib] = t[nn*k + i];
        }
        for (int jb = 0; jb < 8; ++jb) {
            int j = jc * 64 + jb * 8 + ja;
            y[jb] = d[nn*k + j];
        }
        for (int ib = 0; ib < 8; ++ib) {
            for (int jb = 0; jb < 8; ++jb) {
                v[ib][jb] = min(v[ib][jb], x[ib] + y[jb]);
            }
        }
    }
    for (int ib = 0; ib < 8; ++ib) {
        for (int jb = 0; jb < 8; ++jb) {
            int i = ic * 64 + ib * 8 + ia;
            int j = jc * 64 + jb * 8 + ja;
            if (i < n && j < n) {
                r[n*i + j] = v[ib][jb];
            }
        }
    }
}


__global__ void add_padding_v2(const float* r, float* d, int n, int nn) {
    int ja = threadIdx.x;
    int i = blockIdx.y;

    float* t = d + nn * nn;

    for (int jb = 0; jb < nn; jb += 64) {
        int j = jb + ja;
        float v = (i < n && j < n) ? r[n*i + j] : HUGE_VALF;
        d[nn*i + j] = v;
        t[nn*j + i] = v;
    }
}


void step_v0(float* r, const float* d, int n) {
    // Allocate memory & copy data to GPU
    float* dGPU = NULL;
    float* rGPU = NULL;

    CHECK(hipMalloc(&dGPU, n * n * sizeof(float)));
    CHECK(hipMalloc(&rGPU, n * n * sizeof(float)));
    CHECK(hipMemcpy(dGPU, d, n * n * sizeof(float), hipMemcpyHostToDevice));

    // Run kernel
    dim3 dimBlock(16, 16);
    dim3 dimGrid(divup(n, dimBlock.x), divup(n, dimBlock.y));
    kernel_v0<<<dimGrid, dimBlock>>>(rGPU, dGPU, n);
    CHECK(hipGetLastError());

    // Copy data back to CPU & release memory
    CHECK(hipMemcpy(r, rGPU, n * n * sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipFree(dGPU));
    CHECK(hipFree(rGPU));
}


void step_v1(float* r, const float* d, int n) {
    float* dGPU = NULL;
    float* rGPU = NULL;

    CHECK(hipMalloc(&dGPU, n * n * sizeof(float)));
    CHECK(hipMalloc(&rGPU, n * n * sizeof(float)));
    CHECK(hipMemcpy(dGPU, d, n * n * sizeof(float), hipMemcpyHostToDevice));

    dim3 dimBlock(16, 16);
    dim3 dimGrid(divup(n, dimBlock.x), divup(n, dimBlock.y));
    kernel_v1<<<dimGrid, dimBlock>>>(rGPU, dGPU, n);
    CHECK(hipGetLastError());

    CHECK(hipMemcpy(r, rGPU, n * n * sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipFree(dGPU));
    CHECK(hipFree(rGPU));
}


void step_v2(float* r, const float* d, int n) {
    int nn = roundup(n, 64);

    float* dGPU = NULL;
    float* rGPU = NULL;

    CHECK(hipMalloc(&dGPU, 2 * nn * nn * sizeof(float)));
    CHECK(hipMalloc(&rGPU, n * n * sizeof(float)));
    CHECK(hipMemcpy(dGPU, d, n * n * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(rGPU, d, n * n * sizeof(float), hipMemcpyHostToDevice));

    {
        dim3 dimBlock(64, 1);
        dim3 dimGrid(1, nn);
        add_padding_v2<<<dimGrid, dimBlock>>>(rGPU, dGPU, n, nn);
        CHECK(hipGetLastError());
    }

    {
        dim3 dimBlock(8, 8);
        dim3 dimGrid(nn / 64, nn / 64);
        kernel_v2<<<dimGrid, dimBlock>>>(rGPU, dGPU, n, nn);
        CHECK(hipGetLastError());
    }

    CHECK(hipMemcpy(r, rGPU, n * n * sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipFree(dGPU));
    CHECK(hipFree(rGPU));
}


void print_header() {
    std::cout << std::setw(12) << "function"
              << std::setw(12) << "iteration"
              << std::setw(12) << "input size"
              << std::setw(12) << "time (μs)"
              << std::endl;
}
void print_row(const char* name, int i, size_t n, double time) {
    std::cout << std::setw(12) << name
              << std::setw(12) << i
              << std::setw(12) << n*n
              << std::setw(12) << (int)(1e6 * time)
              << std::endl;
}


struct FunctionData {
    const char* name;
    void (*callable)(float*, const float*, int);
    const size_t n;
};


int main(int argc, char** argv) {
    int iterations = 1;
    if (argc > 1) {
        iterations = std::stoi(argv[1]);
    }

    std::vector<FunctionData> functions = {
        {"step_v0", step_v0, BLOCKSIZE << 7},
        {"step_v1", step_v1, BLOCKSIZE << 7},
        {"step_v2", step_v2, BLOCKSIZE << 7},
    };

    print_header();

    for (auto func : functions) {
        const size_t n = func.n;
        for (auto i = 0; i < iterations; ++i) {
            std::vector<float> data(n*n);
            std::generate(data.begin(), data.end(), next_float);
            std::vector<float> result(n*n);
            const auto time_start = std::chrono::high_resolution_clock::now();
            func.callable(result.data(), data.data(), n);
            const auto time_end = std::chrono::high_resolution_clock::now();
            const std::chrono::duration<float> delta_seconds = time_end - time_start;
            print_row(func.name, i + 1, n, delta_seconds.count());
        }
    }

}
